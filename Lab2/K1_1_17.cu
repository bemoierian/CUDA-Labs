// Kernel 1: Each thread computes one element of C


#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void MatAdd(float *A, float *B, float *C, int n)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int j = blockIdx.y * blockDim.y + threadIdx.y;
    if (i < n && j < n)
        C[i * n + j] = A[i * n + j] + B[i * n + j];
}

int main(int argc, char *argv[])
{
    // read testcases from input file
    FILE *in_fptr;
    in_fptr = fopen(argv[1], "r");
    // Create output file to write the results
    FILE *out_fptr;
    out_fptr = fopen(argv[2], "w");
    // Read the number of testcases
    int numOfTestcases;
    fscanf(in_fptr, "%d", &numOfTestcases);
    for (int i = 0; i < numOfTestcases; i++)
    {
        int rows, columns;
        fscanf(in_fptr, "%d %d", &rows, &columns);
        // Size of matrices
        int size = rows * columns * sizeof(float);
        // Matrix initialization
        float *h_A = (float *)malloc(size);
        float *h_B = (float *)malloc(size);
        float *h_C = (float *)malloc(size);
        for (int j = 0; j < rows; j++)
        {
            for (int k = 0; k < columns; k++)
            {
                fscanf(in_fptr, "%f", &h_A[j * columns + k]);
                // printf("Index: %d ", j * columns + k);
                // printf("Value: %f\n", h_A[j * columns + k]);
            }
        }
        for (int j = 0; j < rows; j++)
        {
            for (int k = 0; k < columns; k++)
            {
                fscanf(in_fptr, "%f", &h_B[j * columns + k]);
            }
        }
        // Print the matrices
        // printf("Matrix A:\n");
        // for (int j = 0; j < rows; j++)
        // {
        //     for (int k = 0; k < columns; k++)
        //     {
        //         printf("%f ", h_A[j * rows + k]);
        //     }
        //     printf("\n");
        // }
        // printf("Matrix B:\n");
        // for (int j = 0; j < rows; j++)
        // {
        //     for (int k = 0; k < columns; k++)
        //     {
        //         printf("%f ", h_B[j * rows + k]);
        //     }
        //     printf("\n");
        // }
        // Device memory allocation
        float *d_A, *d_B, *d_C;
        hipMalloc((void **)&d_A, size);
        hipMalloc((void **)&d_B, size);
        hipMalloc((void **)&d_C, size);

        // Data transfer: Host to Device
        hipMemcpy(d_A, h_A, size, hipMemcpyHostToDevice);
        hipMemcpy(d_B, h_B, size, hipMemcpyHostToDevice);

        // Kernel invocation
        dim3 threadsPerBlock(16, 16);
        dim3 numBlocks(((rows - 1) / threadsPerBlock.x + 1), (columns - 1) / threadsPerBlock.y + 1);
        MatAdd<<<numBlocks, threadsPerBlock>>>(d_A, d_B, d_C, rows * columns);

        // Data transfer: Device to Host
        hipMemcpy(h_C, d_C, size, hipMemcpyDeviceToHost);

        // Free device memory
        hipFree(d_A);
        hipFree(d_B);
        hipFree(d_C);

        // Append the result to the output file
        for (int j = 0; j < rows; j++)
        {
            for (int k = 0; k < columns; k++)
            {
                fprintf(out_fptr, "%.1f ", h_C[j * columns + k]);
            }
            fprintf(out_fptr, "\n");
        }

        // Free host memory
        free(h_A);
        free(h_B);
        free(h_C);
    }
    fclose(in_fptr);
    fclose(out_fptr);

    return 0;
}
