// Kernel 2: Each thread computes one row of the output matrix


#include <hip/hip_runtime.h>
#include <stdio.h>

#define N 10000

__global__ void MatAdd(float *A, float *B, float *C, int n)
{
    int row = blockIdx.x * blockDim.x + threadIdx.x;
    if (row < n)
    {
        for (int col = 0; col < n; ++col)
        {
            C[row * n + col] = A[row * n + col] + B[row * n + col];
        }
    }
}

int main(int argc, char *argv[])
{
    // read testcases from input file
    FILE *in_fptr;
    in_fptr = fopen(argv[1], "r");
    // Create output file to write the results
    FILE *out_fptr;
    out_fptr = fopen(argv[2], "w");
    // Read the number of testcases
    int numOfTestcases;
    fscanf(in_fptr, "%d", &numOfTestcases);
    for (int i = 0; i < numOfTestcases; i++)
    {
        int rows, columns;
        fscanf(in_fptr, "%d %d", &rows, &columns);
        // Size of matrices
        int size = rows * columns * sizeof(float);
        // Matrix initialization
        float *h_A = (float *)malloc(size);
        float *h_B = (float *)malloc(size);
        float *h_C = (float *)malloc(size);
        for (int j = 0; j < rows; j++)
        {
            for (int k = 0; k < columns; k++)
            {
                fscanf(in_fptr, "%f", &h_A[j * rows + k]);
            }
        }
        for (int j = 0; j < rows; j++)
        {
            for (int k = 0; k < columns; k++)
            {
                fscanf(in_fptr, "%f", &h_B[j * rows + k]);
            }
        }
        // Device memory allocation
        float *d_A, *d_B, *d_C;
        hipMalloc((void **)&d_A, size);
        hipMalloc((void **)&d_B, size);
        hipMalloc((void **)&d_C, size);

        // Data transfer: Host to Device
        hipMemcpy(d_A, h_A, size, hipMemcpyHostToDevice);
        hipMemcpy(d_B, h_B, size, hipMemcpyHostToDevice);

        // Kernel invocation
        dim3 threadsPerBlock(16);
        dim3 numBlocks((rows * columns + threadsPerBlock.x - 1) / threadsPerBlock.x);
        MatAdd<<<numBlocks, threadsPerBlock>>>(d_A, d_B, d_C, rows * columns);

        // Data transfer: Device to Host
        hipMemcpy(h_C, d_C, size, hipMemcpyDeviceToHost);

        // Free device memory
        hipFree(d_A);
        hipFree(d_B);
        hipFree(d_C);

        // Append the result to the output file
        for (int j = 0; j < rows; j++)
        {
            for (int k = 0; k < columns; k++)
            {
                fprintf(out_fptr, "%f ", h_C[j * rows + k]);
            }
            fprintf(out_fptr, "\n");
        }

        // Free host memory
        free(h_A);
        free(h_B);
        free(h_C);
    }
    fclose(in_fptr);
    fclose(out_fptr);

    return 0;
}
